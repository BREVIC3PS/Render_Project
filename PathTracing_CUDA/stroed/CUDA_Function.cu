#include "hip/hip_runtime.h"
#pragma once
#include "Bounds3.hpp"


#include <vector>
#include "Vector.hpp"
#include "Object.hpp"
#include "Light.hpp"
#include "AreaLight.hpp"
#include "BVH.hpp"
#include "Ray.hpp"
#include "Scene.hpp"

__device__ bool Bounds3::IntersectP(const Ray& ray, const Vector3f& invDir,
                                const std::array<int, 3>& dirIsNeg) const
{
    // invDir: ray direction(x,y,z), invDir=(1.0/x,1.0/y,1.0/z), use this because Multiply is faster that Division
    // dirIsNeg: ray direction(x,y,z), dirIsNeg=[int(x>0),int(y>0),int(z>0)], use this to simplify your logic
    // TODO test if ray bound intersects

    Vector3f tmin = (pMin - ray.origin) * invDir;
    Vector3f tmax = (pMax - ray.origin) * invDir;
    if (dirIsNeg[0])
        std::swap(tmin.x, tmax.x);
    if (dirIsNeg[1])
        std::swap(tmin.y, tmax.y);
    if (dirIsNeg[2])
        std::swap(tmin.z, tmax.z);
    float texit = std::min(tmax.x, std::min(tmax.y, tmax.z));
    float tenter = std::max(tmin.x, std::max(tmin.y, tmin.z));
    return tenter <= texit && texit >= 0;
    

}







// Compute refraction direction using Snell's law
//
// We need to handle with care the two possible situations:
//
//    - When the ray is inside the object
//
//    - When the ray is outside.
//
// If the ray is outside, you need to make cosi positive cosi = -N.I
//
// If the ray is inside, you need to invert the refractive indices and negate the normal N
__device__  Vector3f Scene::refract(const Vector3f &I, const Vector3f &N, const float &ior) const
    {
        float cosi = clamp(-1, 1, dotProduct(I, N));
        float etai = 1, etat = ior;
        Vector3f n = N;
        if (cosi < 0) { cosi = -cosi; } else { std::swap(etai, etat); n= -N; }
        float eta = etai / etat;
        float k = 1 - eta * eta * (1 - cosi * cosi);
        return k < 0 ? 0 : eta * I + (eta * cosi - sqrtf(k)) * n;
    }



    // Compute Fresnel equation
//
// \param I is the incident view direction
//
// \param N is the normal at the intersection point
//
// \param ior is the material refractive index
//
// \param[out] kr is the amount of light reflected
__device__  void Scene::fresnel(const Vector3f &I, const Vector3f &N, const float &ior, float &kr) const
    {
        float cosi = clamp(-1, 1, dotProduct(I, N));
        float etai = 1, etat = ior;
        if (cosi > 0) {  std::swap(etai, etat); }
        // Compute sini using Snell's law
        float sint = etai / etat * sqrtf(std::max(0.f, 1 - cosi * cosi));
        // Total internal reflection
        if (sint >= 1) {
            kr = 1;
        }
        else {
            float cost = sqrtf(std::max(0.f, 1 - sint * sint));
            cosi = fabsf(cosi);
            float Rs = ((etat * cosi) - (etai * cost)) / ((etat * cosi) + (etai * cost));
            float Rp = ((etai * cosi) - (etat * cost)) / ((etai * cosi) + (etat * cost));
            kr = (Rs * Rs + Rp * Rp) / 2;
        }
        // As a consequence of the conservation of energy, transmittance is given by:
        // kt = 1 - kr;
    }


Intersection Scene::intersect(const Ray &ray) const
{
    return this->bvh->Intersect(ray);
}

__device__ void Scene::sampleLight(Intersection &pos, float &pdf) const
{
    float emit_area_sum = 0;
    for (uint32_t k = 0; k < objects.size(); ++k) {
        if (objects[k]->hasEmit()){
            emit_area_sum += objects[k]->getArea();
        }
    }
    float p = get_random_float() * emit_area_sum;
    emit_area_sum = 0;
    for (uint32_t k = 0; k < objects.size(); ++k) {
        if (objects[k]->hasEmit()){
            emit_area_sum += objects[k]->getArea();
            if (p <= emit_area_sum){
                objects[k]->Sample(pos, pdf);
                break;
            }
        }
    }
}

__device__ bool Scene::trace(
        const Ray &ray,
        const std::vector<Object*> &objects,
        float &tNear, uint32_t &index, Object **hitObject)
{
    *hitObject = nullptr;
    for (uint32_t k = 0; k < objects.size(); ++k) {
        float tNearK = kInfinity;
        uint32_t indexK;
        Vector2f uvK;
        if (objects[k]->intersect(ray, tNearK, indexK) && tNearK < tNear) {
            *hitObject = objects[k];
            tNear = tNearK;
            index = indexK;
        }
    }


    return (*hitObject != nullptr);
}

// Implementation of Path Tracing
__device__ Vector3f Scene::castRay(const Ray& ray, int depth) const
{
    // TO DO Implement Path Tracing Algorithm here
    Vector3f hitColor = this->backgroundColor;
    Intersection shade_point_inter = Scene::intersect(ray);
    if (shade_point_inter.happened)
    {
        Vector3f p = shade_point_inter.coords;
        Vector3f wo = ray.direction;
        Vector3f N = shade_point_inter.normal;
        Vector3f L_dir(0), L_indir(0);
        Vector3f p_deviation = (dotProduct(ray.direction, N) < 0) ?
            p + N * EPSILON :
            p - N * EPSILON;

        switch (shade_point_inter.m->getType())
        {
        case MIRROR:
        {
            //Test Russian Roulette with probability RussianRouolette
            float ksi = get_random_float();
            if (ksi < RussianRoulette)
            {
                //wi=sample(wo,N)
                Vector3f wi = normalize(shade_point_inter.m->sample(wo, N));
                //Trace a ray r(p,wi)
                Ray ray_pTowi(p_deviation, wi);
                //If ray r hit a object at q
                Intersection bounce_point_inter = Scene::intersect(ray_pTowi);
                if (bounce_point_inter.happened)
                {
                    float pdf = shade_point_inter.m->pdf(wo, wi, N);
                    if (pdf > EPSILON)
                        L_indir = castRay(ray_pTowi, depth + 1) * shade_point_inter.m->eval(wo, wi, N) * dotProduct(wi, N) / (pdf * RussianRoulette);
                }
            }
            break;
        }
        default:
        {
            //sampleLight(inter,pdf_light)
            Intersection light_point_inter;
            float pdf_light;
            sampleLight(light_point_inter, pdf_light);
            //Get x,ws,NN,emit from inter
            Vector3f x = light_point_inter.coords;
            Vector3f ws = normalize(x - p);
            Vector3f NN = light_point_inter.normal;
            Vector3f emit = light_point_inter.emit;
            float distance_pTox = (x - p).norm();
            //Shoot a ray from p to x
            Ray ray_pTox(p_deviation, ws);
            //If the ray is not blocked in the middleff
            Intersection blocked_point_inter = Scene::intersect(ray_pTox);
            if (abs(distance_pTox - blocked_point_inter.distance < 0.01))
            {
                L_dir = emit * shade_point_inter.m->eval(wo, ws, N) * dotProduct(ws, N) * dotProduct(-ws, NN) / (distance_pTox * distance_pTox * pdf_light);
            }
            //Test Russian Roulette with probability RussianRouolette
            float ksi = get_random_float();
            if (ksi < RussianRoulette)
            {
                //wi=sample(wo,N)
                Vector3f wi = normalize(shade_point_inter.m->sample(wo, N));
                //Trace a ray r(p,wi)
                Ray ray_pTowi(p_deviation, wi);
                //If ray r hit a non-emitting object at q
                Intersection bounce_point_inter = Scene::intersect(ray_pTowi);
                if (bounce_point_inter.happened && !bounce_point_inter.m->hasEmission())
                {
                    float pdf = shade_point_inter.m->pdf(wo, wi, N);
                    if (pdf > EPSILON)
                        L_indir = castRay(ray_pTowi, depth + 1) * shade_point_inter.m->eval(wo, wi, N) * dotProduct(wi, N) / (pdf * RussianRoulette);
                }
            }
            break;
        }
        }
        hitColor = shade_point_inter.m->getEmission() + L_dir + L_indir;
        hitColor.x = (clamp(0, 1, hitColor.x));
        hitColor.y = (clamp(0, 1, hitColor.y));
        hitColor.z = (clamp(0, 1, hitColor.z));

    }
    return hitColor;
}
